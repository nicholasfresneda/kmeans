#include "hip/hip_runtime.h"
#include "Cluster.hpp"
#include <math.h>
#include <fstream>
#include <string>
#include <iostream>

#define MAX_PT 800
using namespace std;


__global__ void updateCoordKernel(long coordSize, int* coordXVals, int* coordYVals, 
                                int* clusterXVals, int* clusterYVals, int kSize,
                                int* coordKMaps)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = tid; i < coordSize; i+= stride)
    {
        int coordX = coordXVals[i];
        int coordY = coordYVals[i];

        //set to first cluster's distance, will be updated in loop
        double p1 = (coordX - clusterXVals[0]) * (coordX - clusterXVals[0]);
        double p2 = (coordY - clusterYVals[0]) * (coordY - clusterYVals[0]);
        int minDist = (int) sqrt(p1 + p2);
        int index = 0;
        for (int j = 1; j < kSize; j++)
        {
            double p1 = (coordX - clusterXVals[j]) * (coordX - clusterXVals[j]);
            double p2 = (coordY - clusterYVals[j]) * (coordY - clusterYVals[j]);
            int newDist = (int) sqrt(p1 + p2);
            if (newDist < minDist)
            {
                minDist = newDist;
                index = j;
            }
        }

        coordKMaps[i] = index;
    }
    
}

Cluster::Cluster(int kSize, long coordSize)
{
    this->kSize = kSize;
    this->coordSize = coordSize;
    //allocate and initialize member variables
    coordXVals = (int*)malloc(coordSize * sizeof(int));
    coordYVals = (int*)malloc(coordSize * sizeof(int));
    coordKMaps = (int*)malloc(coordSize * sizeof(int));
    clusterXVals = (int*)malloc(kSize * sizeof(int));
    clusterYVals = (int*)malloc(kSize * sizeof(int));
}

Cluster::~Cluster()
{
}

void Cluster::writeToDatafile(int fileNum)
{
    //iterate through map
    //writing each coordinates x and y and then cluster num to file
    ofstream myfile;
    std::string file("data" + to_string(fileNum) + ".dat" );

    myfile.open (file);
    for(unsigned int i = 0; i < coordSize; i++)
    {
        myfile << coordXVals[i] << "\t" << coordYVals[i] << "\t"
         << coordKMaps[i] << "\n";
    }

    myfile.close();
    std::string kfile("data" + to_string(fileNum) + "a.dat");
    myfile.open(kfile);
    for (unsigned int i = 0; i < kSize; i++)
    {
        //output each cluster with -1 as third number, to tell gnuplot to color the points the same
        myfile << clusterXVals[i] << "\t" << clusterYVals[i] << "\t" << -1 << "\n";
    }

    myfile.close();
}

void Cluster::updateCoordMap()
{
    //iterate through map
    //calculate closest cluster for each coordinate and update k num in map
    
    
    static bool firstTime = 1;

    if (firstTime)
    {
        hipMalloc(&d_coordXVals, sizeof(int) * coordSize);
        hipMalloc(&d_coordYVals, sizeof(int) * coordSize);
        hipMalloc(&d_coordKMaps, sizeof(int) * coordSize);
        hipMalloc(&d_clusterXVals, sizeof(int) * kSize);
        hipMalloc(&d_clusterYVals, sizeof(int) * kSize);
        firstTime = 0;
        srand(time(NULL));
        for (int i = 0; i < coordSize; i++)
        {
            coordXVals[i] = rand() % MAX_PT;
            coordYVals[i] = rand() % MAX_PT;
            
            //set k cluster for each coord to 0, will be changed later
            coordKMaps[i] = 2;
        }

        for(int i = 0; i < kSize; i++)
        {
            clusterXVals[i] = rand() % MAX_PT;
            clusterYVals[i] = rand() % MAX_PT;
        }

        hipMemcpy(d_coordXVals, coordXVals, coordSize * sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_coordYVals, coordYVals, coordSize * sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_coordKMaps, coordKMaps, coordSize * sizeof(int), hipMemcpyHostToDevice);
    }

    hipMemcpy(d_clusterXVals, clusterXVals, kSize * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_clusterYVals, clusterYVals, kSize * sizeof(int), hipMemcpyHostToDevice);

    int blockSize = 512;
    int numBlocks = (coordSize + blockSize - 1) / blockSize;
    updateCoordKernel<<<numBlocks, blockSize>>>(coordSize, d_coordXVals, d_coordYVals,
                                            d_clusterXVals, d_clusterYVals, kSize, d_coordKMaps);

    hipDeviceSynchronize();
    hipMemcpy(coordKMaps, d_coordKMaps, coordSize * sizeof(int), hipMemcpyDeviceToHost);


}

void Cluster::updateClusterCoords()
{
    //iterate through k cluster vec
    //update cluster with average of each point in maps 
    
    //copy cluster to compare for later
    int copyXCluster[kSize];
    int copyYCluster[kSize];
    for (int i = 0; i < kSize; i++)
    {
        copyXCluster[i] = clusterXVals[i];
        copyYCluster[i] = clusterYVals[i];
    }
    for (unsigned int i = 0; i < kSize; i++)
    {
        int sumX = 0;
        int sumY = 0;
        int count = 0;
        for (unsigned int j = 0; j < coordSize; j++)
        {
            if ((unsigned int) coordKMaps[j] == i)
            {
                count++;
                sumX += coordXVals[j];
                sumY += coordYVals[j];
            }
        }

        int newXVal = 0;
        int newYVal = 0;
        if (count != 0)
        {
           newXVal = sumX / count;
           newYVal = sumY / count;
        }

        clusterXVals[i] = newXVal;
        clusterYVals[i] = newYVal;
    }
    
    bool converged = true;
    for (unsigned int i = 0; i < kSize; i++)
    {
        if (copyXCluster[i] != clusterXVals[i]
         || copyYCluster[i] != clusterYVals[i])
        {
            converged = false;
            break;
        }
    }

    this->converged = converged;
}

