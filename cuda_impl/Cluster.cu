#include "hip/hip_runtime.h"
#include "Cluster.hpp"
#include <math.h>
#include <fstream>
#include <string>
#include <iostream>

#define MAX_PT 800
using namespace std;


__global__ void updateCoordKernel(long coordSize, int* coordXVals, int* coordYVals, 
                                int* clusterXVals, int* clusterYVals, int kSize,
                                int* coordKMaps)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = tid; i < coordSize; i+= stride)
    {
        int coordX = coordXVals[i];
        int coordY = coordYVals[i];

        //set to first cluster's distance, will be updated in loop
        double p1 = (coordX - clusterXVals[0]) * (coordX - clusterXVals[0]);
        double p2 = (coordY - clusterYVals[0]) * (coordY - clusterYVals[0]);
        int minDist = (int) sqrt(p1 + p2);
        int index = 0;
        for (int j = 1; j < kSize; j++)
        {
            double p1 = (coordX - clusterXVals[j]) * (coordX - clusterXVals[j]);
            double p2 = (coordY - clusterYVals[j]) * (coordY - clusterYVals[j]);
            int newDist = (int) sqrt(p1 + p2);
            if (newDist < minDist)
            {
                minDist = newDist;
                index = j;
            }
        }

        coordKMaps[i] = index;
    }
    
}

Cluster::Cluster(int kSize, long coordSize)
{
    this->kSize = kSize;
    this->coordSize = coordSize;
    //allocate and initialize member variables
    coordXVals = new int(coordSize);
    coordYVals = new int(coordSize);
    coordKMaps = new int(coordSize);
    clusterXVals = new int(kSize);
    clusterYVals = new int(kSize);
}

Cluster::~Cluster()
{
}

void Cluster::writeToDatafile(int fileNum)
{
    //iterate through map
    //writing each coordinates x and y and then cluster num to file
    ofstream myfile;
    std::string file("data" + to_string(fileNum) + ".dat" );

    myfile.open (file);
    for(unsigned int i = 0; i < coordSize; i++)
    {
        myfile << coordXVals[i] << "\t" << coordYVals[i] << "\t"
         << coordKMaps[i] << "\n";
    }

    myfile.close();
    std::string kfile("data" + to_string(fileNum) + "a.dat");
    myfile.open(kfile);
    for (unsigned int i = 0; i < kSize; i++)
    {
        //output each cluster with -1 as third number, to tell gnuplot to color the points the same
        myfile << clusterXVals[i] << "\t" << clusterYVals[i] << "\t" << -1 << "\n";
    }

    myfile.close();
}

void Cluster::updateCoordMap()
{
    //iterate through map
    //calculate closest cluster for each coordinate and update k num in map
    
    
    static bool firstTime = 1;

    if (firstTime)
    {
        hipMallocManaged(&coordXVals, sizeof(int) * coordSize);
        hipMallocManaged(&coordYVals, sizeof(int) * coordSize);
        hipMallocManaged(&coordKMaps, sizeof(int) * coordSize);
        hipMallocManaged(&clusterXVals, sizeof(int) * kSize);
        hipMallocManaged(&clusterYVals, sizeof(int) * kSize);
        firstTime = 0;
        srand(time(NULL));
        for (int i = 0; i < coordSize; i++)
        {
            coordXVals[i] = rand() % MAX_PT;
            coordYVals[i] = rand() % MAX_PT;
            
            //set k cluster for each coord to 0, will be changed later
            coordKMaps[i] = 0;
        }

        for(int i = 0; i < kSize; i++)
        {
            clusterXVals[i] = rand() % MAX_PT;
            clusterYVals[i] = rand() % MAX_PT;
        }
    }
    

    int blockSize = 256;
    int numBlocks = (coordSize + blockSize - 1) / blockSize;
    updateCoordKernel<<<numBlocks, blockSize>>>(coordSize, coordXVals, coordYVals,
                                            clusterXVals, clusterYVals, kSize, coordKMaps);

    hipDeviceSynchronize();
}

void Cluster::updateClusterCoords()
{
    //iterate through k cluster vec
    //update cluster with average of each point in maps 
    
    //copy cluster to compare for later
    int copyXCluster[kSize];
    int copyYCluster[kSize];
    for (int i = 0; i < kSize; i++)
    {
        copyXCluster[i] = clusterXVals[i];
        copyYCluster[i] = clusterYVals[i];
    }
    for (unsigned int i = 0; i < kSize; i++)
    {
        int sumX = 0;
        int sumY = 0;
        int count = 0;
        for (unsigned int j = 0; j < coordSize; j++)
        {
            if ((unsigned int) coordKMaps[j] == i)
            {
                count++;
                sumX += coordXVals[j];
                sumY += coordYVals[j];
            }
        }

        int newXVal = 0;
        int newYVal = 0;
        if (count != 0)
        {
           newXVal = sumX / count;
           newYVal = sumY / count;
        }

        clusterXVals[i] = newXVal;
        clusterYVals[i] = newYVal;
    }
    
    bool converged = true;
    for (unsigned int i = 0; i < kSize; i++)
    {
        if (copyXCluster[i] != clusterXVals[i]
         || copyYCluster[i] != clusterYVals[i])
        {
            converged = false;
            break;
        }
    }

    this->converged = converged;
}

